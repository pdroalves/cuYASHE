#include "hip/hip_runtime.h"
/**
 * cuYASHE
 * Copyright (C) 2015-2016 cuYASHE Authors
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <NTL/ZZ_pEX.h>

#include "polynomial.h"
#include "ciphertext.h"
#include "yashe.h"
#include "settings.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

NTL_CLIENT


#define BILLION  1000000000L
#define MILLION  1000000L
#define N 100

double compute_time_ms(struct timespec start,struct timespec stop){
  return (( stop.tv_sec - start.tv_sec )*BILLION + ( stop.tv_nsec - start.tv_nsec ))/MILLION;
}

uint64_t get_cycles() {
  unsigned int hi, lo;
  asm (
    "cpuid\n\t"/*serialize*/
    "rdtsc\n\t"/*read the clock*/
    "mov %%edx, %0\n\t"
    "mov %%eax, %1\n\t" 
    : "=r" (hi), "=r" (lo):: "%rax", "%rbx", "%rcx", "%rdx"
  );
  return ((uint64_t) lo) | (((uint64_t) hi) << 32);
}


int main(void){
    uint64_t t;
    Yashe cipher;
    int degree;
    Polynomial phi;
    ZZ_pX NTL_Phi;

    srand (36251);

    // Params
    ZZ q;
    q = conv<ZZ>("1171313591017775093490277364417L");
    // q = conv<ZZ>("655615111");
    Polynomial::global_mod = q;
    ZZ_p::init(q); // Defines GF(q)

    t = 35951;
    degree = 4096;
    int w = 72;

    Polynomial::BuildNthCyclotomic(&phi, degree); // generate an cyclotomic polynomial
    phi.set_mod(Polynomial::global_mod);
    Polynomial::global_phi = &phi;

    // Set params to NTL (just for comparison reasons)
    ZZ_p::init(Polynomial::global_mod);
    for(int i = 0; i <= phi.deg();i++){
      NTL::SetCoeff(NTL_Phi,i,conv<ZZ_p>(phi.get_coeff(i)));
    }
    ZZ_pE::init(NTL_Phi);

    CUDAFunctions::init(2*degree);

    Polynomial::gen_crt_primes(Polynomial::global_mod,degree);

    // Yashe
    cipher = Yashe();

    Yashe::d = degree;
    Yashe::phi = phi;
    Yashe::q = q;
    // std::cout << ZZ_p::modulus() << std::endl;
    // std::cout << q << std::endl;

    Yashe::t = t;
    Yashe::w = w;
    Yashe::lwq = floor(NTL::log(q)/(log(2)*w)+1);

    cipher.generate_keys();

    //////////
    struct timespec start, stop;

    Ciphertext cA;
    Ciphertext cB;
    Polynomial pA;
    Polynomial pB;

    Ciphertext::random(&cA,degree-1);
    Ciphertext::random(&cB,degree-1);
    Polynomial::random(&pA,degree-1);
    Polynomial::random(&pB,degree-1);
    
    cA.update_device_data();
    cB.update_device_data();

    std::cout << "Starting..." << std::endl << std::endl;
    clock_gettime( CLOCK_REALTIME, &start);
    for(unsigned int i = 0; i < 100;i++){
      Ciphertext c = cA+cB;
      hipError_t result = hipDeviceSynchronize();
      assert(result == hipSuccess);
    }
    clock_gettime( CLOCK_REALTIME, &stop);
    std::cout << "Ciphertext sum: " << compute_time_ms(start,stop)/100 << std::endl;


    clock_gettime( CLOCK_REALTIME, &start);
    for(unsigned int i = 0; i < 100;i++){
      cA += cB;
      hipError_t result = hipDeviceSynchronize();
      assert(result == hipSuccess);
    }
    clock_gettime( CLOCK_REALTIME, &stop);
    std::cout << "Ciphertext inplace sum: " << compute_time_ms(start,stop)/100 << std::endl;


    pA.update_device_data();
    pB.update_device_data();

    clock_gettime( CLOCK_REALTIME, &start);
    for(unsigned int i = 0; i < 100;i++){
      Polynomial p = pA+pB;
      hipError_t result = hipDeviceSynchronize();
      assert(result == hipSuccess);
    }
    clock_gettime( CLOCK_REALTIME, &stop);
    std::cout << "Polynomial sum: " << compute_time_ms(start,stop)/100 << std::endl;


    clock_gettime( CLOCK_REALTIME, &start);
    for(unsigned int i = 0; i < 100;i++){
      pA += pB;
      hipError_t result = hipDeviceSynchronize();
      assert(result == hipSuccess);
    }
    clock_gettime( CLOCK_REALTIME, &stop);
    std::cout << "Polynomial inplace sum: " << compute_time_ms(start,stop)/100 << std::endl;

}
