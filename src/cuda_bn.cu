#include "hip/hip_runtime.h"
/**
 * cuYASHE
 * Copyright (C) 2015-2016 cuYASHE Authors
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "cuda_bn.h"
#include "settings.h"
#include "cuda_functions.h"
#include "polynomial.h"

__constant__ cuyasheint_t CRTPrimesConstant[PRIMES_BUCKET_SIZE];

__constant__ cuyasheint_t M[STD_BNT_WORDS_ALLOC];
__constant__ int M_used;
__constant__ cuyasheint_t u[STD_BNT_WORDS_ALLOC];
__constant__ int u_used;
__constant__ cuyasheint_t Mpis[STD_BNT_WORDS_ALLOC*PRIMES_BUCKET_SIZE];
__constant__ int Mpis_used[PRIMES_BUCKET_SIZE];
__constant__ cuyasheint_t invMpis[PRIMES_BUCKET_SIZE];

////////////////////////
// Auxiliar functions //
////////////////////////

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__host__ __device__ int max_d(int a,int b){
	return (a >= b)*a + (b > a)*b;
}

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__host__ __device__ int min_d(int a,int b){
	return (a <= b)*a + (b < a)*b;
}

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__device__ void swap_d(bn_t *a,bn_t *b){
	bn_t tmp = *a;
	*a = *b;
	*b = tmp;
}

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__device__ int isZero(int x) {
    unsigned zero;
    zero = x;
    zero = 1 ^ ((zero | -zero) >> 31) & 1;
    return zero;    
}

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__device__ int isNotZero(int x){
	unsigned result = isZero(x);
	return isZero(result);
}

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__host__ __device__ uint64_t  lessThan(uint64_t x, uint64_t y) {    
    uint64_t less;    
    less = x-y;
    less >>= sizeof(uint64_t)*8-1;    
    return less;        
}

// /**
//  * [max_d description]
//  * @param  a [description]
//  * @param  b [description]
//  * @return   [description]
//  */
// __device__ unsigned greaterOrEqualThan(int x, int y){
// 	return lessThan(y,x);
// }

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__device__ unsigned isEqual(int x, int y) {    
    unsigned equal;    
    equal = x-y; // "equal" turns 0 if x = y    
    equal = 1 ^ ((equal | -equal) >> 31) & 1; // "equal" turns 1 iff enable was 0
    return equal;    
}

/**
 * [max_d description]
 * @param  a [description]
 * @param  b [description]
 * @return   [description]
 */
__host__ __device__ void dv_zero(cuyasheint_t *a, int digits) {
	int i;
 
	// if (digits > DV_DIGS) {
	// 	std::cout << "ERR_NO_VALID" << std::endl;
	// 	exit(1);
	// }	
	for (i = 0; i < digits; i++, a++)
		(*a) = 0;

	return;
}

/**
 * Returns the highest bit set on a digit.
 *
 * About __builtin_clzll: https://gcc.gnu.org/onlinedocs/gcc/Other-Builtins.html
 * @param  a [description]
 * @return   [description]
 */
__device__ int util_bits_dig(cuyasheint_t a) {
	return WORD - __clz(a);
}

/**
 * Set a big number struct to zero
 * @param a operand
 */
__host__ __device__ void bn_zero(bn_t *a) {
	a->sign = BN_POS;
	a->used = 0;
	dv_zero(a->dp, a->alloc);
}

__host__ __device__ void bn_zero_non_used(bn_t *a) {
	dv_zero(a->dp+a->used, a->alloc-a->used);
}

__host__ __device__ bool bn_is_zero(const bn_t* a) {
	#ifdef __CUDA_ARCH__
	/**
	 * This version doesn't have branchings
	 */
		// return !isEqual( isEqual(a->used,0) + isEqual(a->used,1)*isEqual(a->dp[0],0)
		// 				,false);
		return a->used == 0;
	 // bool testA = a->used == 0;
	 // bool testB = (a->used == 1) && (a->dp[0] == 0);
	 // return testA || testB;
	#else
		if (a->used == 0) {
			return true;
		}
		if ((a->used == 1) && (a->dp[0] == 0)) {
			return true;
		}
		return false;
	#endif
}

__global__ void bn_get_deg(int *r, bn_t *coefs, int N){
	/**
	 * This kernel must be executed by N threads
	 */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if(tid < N){
		coefs[tid].used = get_used_index(coefs[tid].dp,coefs[tid].alloc)+1;
		r[tid] = !bn_is_zero(&coefs[tid]);		
	}
}

__host__ int callBNGetDeg(bn_t *coefs, int N){
	/**
	 * Alloc memory
	 */
	int *d_result;
	int *h_result;
	hipError_t result = hipMalloc((void**)&d_result,N*sizeof(int));
	assert(result == hipSuccess);
	h_result = (int*)malloc(N*sizeof(int));
	
	/** 
	 * Kernel
	 */
	int size = N;
	const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);

	bn_get_deg<<<gridDim,blockDim>>>(d_result,coefs,N);
	result = hipGetLastError();
	assert(result == hipSuccess);

	/** 
	 * Recover
	 */
	result = hipMemcpy(h_result,d_result,N*sizeof(int),hipMemcpyDeviceToHost);
	assert(result == hipSuccess);

	for(int i = N-1; i >= 0; i--)
		if(h_result[i] != 0)
			return i;
	return -1;
}

/**
 * Set a big number to digit
 * @param a     input: big number
 * @param digit input: digit
 */
__host__ __device__ void bn_set_dig(bn_t *a, cuyasheint_t digit) {
	bn_zero(a);	
	a->dp[0] = digit;
	a->used = 1;
	a->sign = BN_POS;
}

__host__ void bn_new(bn_t *a){
  a->used = 0;
  a->alloc = STD_BNT_WORDS_ALLOC;
  a->sign = BN_POS;
  // std::cout << "Will alloc " << (a->alloc*sizeof(cuyasheint_t)) << " bytes" << std::endl;
  a->dp = (cuyasheint_t*) malloc(a->alloc*sizeof(cuyasheint_t));
}

// __device__ void bn_new_d(bn_t *a){
//   a->used = 0;
//   a->alloc = STD_BNT_WORDS_ALLOC;
//   a->sign = BN_POS;
//   hipMalloc(&a->dp,a->alloc*sizeof(cuyasheint_t));
// }

__host__ void bn_free(bn_t *a){
  if(a->dp != NULL && a->alloc > 0){
	hipError_t result = hipFree((a->dp));
	if(result != hipSuccess){
		std::cout << hipGetErrorString(result) << std::endl;
		hipGetLastError();//Reset
	}
  	// assert(result == hipSuccess);
  }

  a->used = 0;
  a->alloc = 0;  

}

/**
 * Compares two digit vectors of the same size.
 *
 * @param  a    [description]
 * @param  b    [description]
 * @param  size [description]
 * @return      [description]
 */
template<typename T>
__host__ __device__ int bn_cmpn_low(const T *a, const T *b, int size) {
	int i, r;

	a += (size - 1);
	b += (size - 1);

	r = CMP_EQ;
	for (i = 0; i < size; i++, --a, --b) {
		// r = 			r				*(!((*a != *b) && r == CMP_EQ)) + 
		// 	((*a > *b)*CMP_GT + (*a <= *b)*CMP_LT)*(*a != *b && r == CMP_EQ);
		if (*a != *b && r == CMP_EQ) {
			r = (*a > *b ? CMP_GT : CMP_LT);
		}
	}
	return r;
}

__host__ __device__ int bn_cmp_abs(const bn_t *a, const bn_t *b) {
	if(a->used != b-> used)
		return CMP_GT*(a->used > b->used) + CMP_LT*(a->used < b->used); 
	else
		return bn_cmpn_low<cuyasheint_t>(a->dp, b->dp, a->used);
}

/**
 * Increase the allocated memory for a bn_t object.
 * @param a        input/output:operand
 * @param new_size input: new_size for dp
 */
__host__ void bn_grow(bn_t *a,const unsigned int new_size){
  // We expect that a->alloc <= new_size
  if((unsigned int)a->alloc > new_size)
  	return;

  std::cout << "Will grow " << (new_size*sizeof(cuyasheint_t)) << " bytes" << std::endl;

  hipMalloc((void**)(&(a->dp)+a->alloc),new_size*sizeof(cuyasheint_t));
  a->alloc = new_size;

}

__host__ __device__ void bn_copy(bn_t *a, bn_t *b){
	// Copy b to a
	assert(a->alloc >= b->alloc);
	a->used = b->used;
	a->sign = b->sign;
	for(int i = 0; i < b->used; i++)
		a->dp[i] = b->dp[i];
}
__host__ __device__ void bn_2_compl(bn_t *a){
	for(int i = 0; i < a->used; i++)
		a->dp[i] = (a->dp[i]^UINT64_MAX);
	a->dp[0] += 1; 
}

__host__ __device__ void bn_bitwise_and(bn_t *a, bn_t *b){
	// Compute a = a & b
	for(int i = 0; i < min_d(a->used,b->used);i++)
		a->dp[i] = (a->dp[i] & b->dp[i]);
}

__host__ __device__ void bn_truncate(bn_t *a, int bits){
	//
	// Set a = a & (2^bits - 1)
	// 
	int index = (a->used*WORD) / bits;
	a->dp[index] = (a->dp[index] << (WORD - bits));  
	a->dp[index] = (a->dp[index] >> (WORD - bits));
	a->used = index+1;
}

/**
 * Shifts a digit vector to the right by some digits. 
 * Computes c = a >> (digits * DIGIT).
 *
 * 64 bits version
 * @param c      [description]
 * @param a      [description]
 * @param size   [description]
 * @param digits [description]
 */
template<typename T>
__host__ __device__ void bn_rshd_low(T *c, const T *a, int size, int digits) {
	const T *top;
	T *bot;
	int i;

	top = a + digits;
	bot = c;

	for (i = 0; i < size - digits; i++, top++, bot++) {
		*bot = *top;
	}
}


/**
 * Shifts a digit vector to the right by an amount smaller than a digit. 
 * Computes c = a >> bits.
 *
 * 64 bits version
 * @param  c    [description]
 * @param  a    [description]
 * @param  size [description]
 * @param  bits [description]
 * @return      [description]
 */
__host__ __device__ uint64_t bn_rshb_low(uint64_t *c, const uint64_t *a, int size, int bits) {
	int i;
	uint64_t r, carry, shift, mask;

	c += size - 1;
	a += size - 1;
	/* Prepare the bit mask. */
	shift = 64 - bits;
	carry = 0;
	mask = MASK(bits);
	for (i = size - 1; i >= 0; i--, a--, c--) {
		/* Get the needed least significant bits. */
		r = (*a) & mask;
		/* Shift left the operand. */
		*c = ((*a) >> bits) | (carry << shift);
		/* Update the carry. */
		carry = r;
	}
	return carry;
}

/**
 * Shifts a digit vector to the right by an amount smaller than a digit. 
 * Computes c = a >> bits.
 *
 * 32 bits version
 * @param  c    [description]
 * @param  a    [description]
 * @param  size [description]
 * @param  bits [description]
 * @return      [description]
 */
__host__ __device__ uint32_t bn_rshb_low_32(uint32_t *c, const uint32_t *a, int size, int bits) {
	int i;
	uint32_t r, carry, shift, mask;

	c += size - 1;
	a += size - 1;
	/* Prepare the bit mask. */
	shift = 32 - bits;
	carry = 0;
	mask = MASK_32(bits);
	for (i = size - 1; i >= 0; i--, a--, c--) {
		/* Get the needed least significant bits. */
		r = (*a) & mask;
		/* Shift left the operand. */
		*c = ((*a) >> bits) | (carry << shift);
		/* Update the carry. */
		carry = r;
	}
	return carry;
}


/**
 * Shifts a digit vector to the left by some digits. 
 * Computes c = a << (digits * DIGIT). 
 *
 * @param  c    [description]
 * @param  a    [description]
 * @param  size [description]
 * @param  bits [description]
 * @return      [description]
 */
template<typename T>
__host__ __device__ void bn_lshd_low(T *c, const T *a, int size, int digits) {
	T *top;
	const T *bot;
	int i;

	top = c + size + digits - 1;
	bot = a + size - 1;

	for (i = 0; i < size; i++, top--, bot--) {
		*top = *bot;
	}
	for (i = 0; i < digits; i++, c++) {
		*c = 0;
	}
}
/**
 * Shifts a digit vector to the left by an amount smaller than a digit. 
 *
 * 64 bits version
 * Computes c = a << bits.
 * @param  c    [description]
 * @param  a    [description]
 * @param  size [description]
 * @param  bits [description]
 * @return      [description]
 */
template<typename T>
__host__ __device__  T bn_lshb_low(T *c, const T *a, int size, int bits);

template<>
__host__ __device__  uint64_t bn_lshb_low<uint64_t>(uint64_t *c, const uint64_t *a, int size, int bits) {
	int i;
	uint64_t r, carry, shift, mask;

	shift = 64 - bits;
	carry = 0;
	mask = MASK(bits);
	for (i = 0; i < size; i++, a++, c++) {
		/* Get the needed least significant bits. */
		r = ((*a) >> shift) & mask;
		/* Shift left the operand. */
		*c = ((*a) << bits) | carry;
		/* Update the carry. */
		carry = r;
	}
	return carry;
}

/**
 * Shifts a digit vector to the left by an amount smaller than a digit. 
 *
 * 32 bits version
 * Computes c = a << bits.
 * @param  c    [description]
 * @param  a    [description]
 * @param  size [description]
 * @param  bits [description]
 * @return      [description]
 */
template<>
__host__ __device__  uint32_t bn_lshb_low<uint32_t>(uint32_t *c, const uint32_t *a, int size, int bits) {
	int i;
	uint32_t r, carry, shift, mask;

	shift = 32 - bits;
	carry = 0;
	mask = MASK_32(bits);
	for (i = 0; i < size; i++, a++, c++) {
		/* Get the needed least significant bits. */
		r = ((*a) >> shift) & mask;
		/* Shift left the operand. */
		*c = ((*a) << bits) | carry;
		/* Update the carry. */
		carry = r;
	}
	return carry;
}


////////////////
// Operators //
//////////////

// Mod
__host__ __device__ cuyasheint_t bn_mod1_low(	const uint64_t *a,
												const int size,
												const uint32_t b) {
	// Computes a % b, where b is a one-word number
	
	uint64_t w;
	uint32_t r;
	int i;

	w = 0;
	for (i = size - 1; i >= 0; i--) {
		// Second 32 bits word
		uint32_t hi = uint32_t(a[i] >> 32);
		w = ((w << ((uint64_t)32)) | ((uint64_t)hi))*(hi > 0) + w*(hi == 0);

		r = (uint32_t)(w/b)*(w >= b);
		w -= (((uint64_t)r) * ((uint64_t)b))*(w >= b || hi > 0);
		
		// First 32 bits word
		uint32_t lo = uint32_t(a[i] & 0xFFFFFFFF);
		w = (w << ((uint64_t)32)) | ((uint64_t)lo);

		r = (uint32_t)(w/b)*(w >= b);
		w -= (((uint64_t)r) * ((uint64_t)b))*(w >= b);
		
	}
	return (cuyasheint_t)w;
}

// Multiply 

/**
 * Computes a*digit
 * @param  c     output: result
 * @param  a     input: many-words first operand
 * @param  digit input: one-word second operand
 * @param  size  input: number of words in a
 * @return       output: result's last word
 */
__host__ __device__ uint64_t bn_mul1_low(uint64_t *c,
											const uint64_t *a,
											uint64_t digit,
											int size) {
	int i;
	uint64_t carry;
	carry = 0;
	for (i = 0; i < size; i++, a++, c++) {	
  		#ifdef __CUDA_ARCH__
		////////////
  		// device //
		////////////
  		
		/**
		 * Multiply the digit *a by b and propagate the carry
		 */
		uint64_t lo = (*a)*digit;
		uint64_t hi = __umul64hi(*a,digit) + (lo + carry < lo);
		lo = lo + carry;

		/* Increment the column and assign the result. */
		*c = lo;
		/* Update the carry. */
		carry = hi;
		#else
		//////////
		// host //
		//////////
		__uint128_t r = (((__uint128_t)(*a)) * ((__uint128_t)digit) );
		*c = (r & 0xffffffffffffffffL);
		carry = (r>>64);
		#endif
	}

	return carry;
}

__host__ __device__ uint32_t bn_mul1_low_32(uint32_t *c,
											const uint32_t *a,
											uint32_t digit,
											int size) {
	int i;
	uint32_t carry;
	uint64_t r;

	carry = 0;
	for (i = 0; i < size; i++, a++, c++) {
		/* Multiply the digit *tmpa by b and accumulate with the previous
		 * result in the same columns and the propagated carry. */
		r = (uint64_t)(carry) + (uint64_t)(*a) * (uint64_t)(digit);
		/* Increment the column and assign the result. */
		*c = (uint32_t)r;
		/* Update the carry. */
		carry = (uint32_t)(r >> (uint64_t)32);
	}
	return carry;
}

/**
 * Computes 64bits a*b mod c
 * @param result       output: result
 * @param a            input: first 64 bits operand
 * @param b            input: second 64 bits operand 
 * @param c 		   input: module
 */
__device__ void bn_64bits_mulmod(cuyasheint_t *result,
									cuyasheint_t a,
									cuyasheint_t b,
									uint32_t m
									){
	/////////
	// Mul //
	/////////
	uint64_t rHi = __umul64hi(a,b);
	uint64_t rLo = a*b;

	/////////
	// Mod //
	/////////
	uint64_t r[] = {	(uint64_t)(rLo & 0xFFFFFFFF),
						(uint64_t)(rLo >> 32),
						(uint64_t)(rHi & 0xFFFFFFFF),
						(uint64_t)(rHi >> 32) 
					};
	*result = bn_mod1_low(r,4,(uint64_t)m);
}

__device__ inline uint64_t div128_64(uint64_t a_hi, uint64_t a_lo, uint64_t b){

	  // quotient
  uint64_t q = a_lo << 1;

  // remainder
  uint64_t rem = a_hi;

  uint64_t carry = a_lo >> 63;
  uint64_t temp_carry = 0;
  int i;

  for(i = 0; i < 64; i++)
  {
    temp_carry = rem >> 63;
    rem <<= 1;
    rem |= carry;
    carry = temp_carry;

    if(carry == 0)
    {
      if(rem >= b)
      {
        carry = 1;
      }
      else
      {
        temp_carry = q >> 63;
        q <<= 1;
        q |= carry;
        carry = temp_carry;
        continue;
      }
    }

    rem -= b;
    rem -= (1 - carry);
    carry = 1;
    temp_carry = q >> 63;
    q <<= 1;
    q |= carry;
    carry = temp_carry;
  }

  return q;
}

// Div
/**
 * Divides a digit vector by another digit vector. 
 * Computes c = floor(a / b) and d = a mod b. 
 * 
 * The dividend and the divisor are destroyed inside the function.
 * 
 * @param c  [description]
 * @param d  [description]
 * @param a  [description]
 * @param sa [description]
 * @param b  [description]
 * @param sb [description]
 */
__device__ void bn_divn_low(cuyasheint_t *c, cuyasheint_t *d, cuyasheint_t *a, int sa, cuyasheint_t *b, int sb) {
	int norm, i, n, t, sd;
	cuyasheint_t carry, t1[3], t2[3];

	if(sa == 0 || sb == 0)
		return;

	/* Normalize x and y so that the leading digit of y is bigger than
	 * 2^(BN_DIGIT-1). */
	norm = util_bits_dig(b[sb - 1]) % BN_DIGIT;

	if (norm < (int)(BN_DIGIT - 1)) {
		norm = (BN_DIGIT - 1) - norm;
		carry = bn_lshb_low(a, a, sa, norm);
		if (carry) {
			a[sa++] = carry;
		}
		carry = bn_lshb_low(b, b, sb, norm);
		if (carry) {
			b[sb++] = carry;
		}
	} else {
		norm = 0;
	}

	n = sa - 1;
	t = sb - 1;

	/* Shift y so that the most significant digit of y is aligned with the
	 * most significant digit of x. */
	bn_lshd_low<cuyasheint_t>(b, b, sb, (n - t)+2);

	/* Find the most significant digit of the quotient. */
	while (bn_cmpn_low(a, b, sa) != CMP_LT) {
		c[n - t]++;
		bn_subn_low(a, a, b, sa);
	}
	/* Shift y back. */

	bn_rshd_low<cuyasheint_t>(b, b, sb + (n - t), (n - t));

	/* Find the remaining digits. */
	for (i = n; i >= (t + 1); i--) {
		if (i > sa) {
			continue;
		}

		if (a[i] == b[t]) {
			c[i - t - 1] = -1;
		} else {
			cuyasheint_t tmp_hi,tmp_low;
			tmp_hi = a[i];
			tmp_low = a[i-1];
			c[i - t - 1] = div128_64(tmp_hi, tmp_low, b[t]);
		}
		c[i - t - 1]++;

		do {
			c[i - t - 1]--;
			t1[0] = (t - 1 < 0) ? 0 : b[t - 1];
			t1[1] = b[t];

			carry = bn_mul1_low(t1, t1, c[i - t - 1], 2);
			t1[2] = carry;

			t2[0] = (i - 2 < 0) ? 0 : a[i - 2];
			t2[1] = (i - 1 < 0) ? 0 : a[i - 1];
			t2[2] = a[i];
		} while (bn_cmpn_low(t1, t2, 3) == CMP_GT);

		carry = bn_mul1_low(d, b, c[i - t - 1], sb);
		sd = sb;
		if (carry) {
			d[sd++] = carry;
		}

		carry = bn_subn_low(a + (i - t - 1), a + (i - t - 1), d, sd);
		sd += (i - t - 1);
		if (sa - sd > 0) {
			carry = bn_sub1_low(a + sd, a + sd, carry, sa - sd);
		}

		if (carry) {
			sd = sb + (i - t - 1);
			carry = bn_addn_low(a + (i - t - 1), a + (i - t - 1), b, sb);
			carry = bn_add1_low(a + sd, a + sd, carry, sa - sd);
			c[i - t - 1]--;
		}
	}
	/* Remainder should be not be longer than the divisor. */
	bn_rshb_low(d, a, sb, norm);
}
// Add

/**
 * Computes a+b
 * @param  c    output: result
 * @param  a    input: many-words first operand
 * @param  b    input: many-words second operand
 * @param  size input: number of words to add
 * @return      output: result's last word
 */
__host__ __device__ uint64_t bn_addn_low(uint64_t *c,
									uint64_t *a,
									uint64_t *b,
									const int size
									) {
	int i;
	register uint64_t carry, c0, c1, r0, r1;

	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		r0 = (*a) + (*b);
		c0 = (r0 < (*a));
		r1 = r0 + carry;
		c1 = (r1 < r0);
		carry = c0 | c1;
		(*c) = r1;
	}
	return carry;
}

__host__ __device__ uint32_t bn_addn_low_32(uint32_t *c,
									uint32_t *a,
									uint32_t *b,
									const int size
									) {
	int i;
	register uint32_t carry, c0, c1, r0, r1;

	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		r0 = (*a) + (*b);
		c0 = (r0 < (*a));
		r1 = r0 + carry;
		c1 = (r1 < r0);
		carry = c0 | c1;
		(*c) = r1;
	}
	return carry;
}

/**
 * [bn_add1_low description]
 * @param  c     [description]
 * @param  a     [description]
 * @param  digit [description]
 * @param  size  [description]
 * @return       [description]
 */
__host__ __device__ uint64_t bn_add1_low(uint64_t *c, const uint64_t *a, uint64_t digit, int size) {
	int i;
	register uint64_t carry, r0;

	carry = digit;
	for (i = 0; i < size && carry; i++, a++, c++) {
		r0 = (*a) + carry;
		carry = (r0 < carry);
		(*c) = r0;
	}
	for (; i < size; i++, a++, c++) {
		(*c) = (*a);
	}
	return carry;
}

__host__ __device__ uint32_t bn_add1_low_32(uint32_t *c, const uint32_t *a, uint32_t digit, int size) {
	int i;
	register uint32_t carry, r0;

	carry = digit;
	for (i = 0; i < size && carry; i++, a++, c++) {
		r0 = (*a) + carry;
		carry = (r0 < carry);
		(*c) = r0;
	}
	for (; i < size; i++, a++, c++) {
		(*c) = (*a);
	}
	return carry;
}

////////////////////////
// Subtract
////////////////////////
/**
 * bn_subn_low computes a-b. If a < b, returns 1. Else returns 0.
 * @param  c    [description]
 * @param  a    [description]
 * @param  b    [description]
 * @param  size [description]
 * @return      [description]
 */
__host__ __device__ uint64_t bn_subn_low(uint64_t * c, const uint64_t * a,
		const uint64_t * b, int size) {
	int i;
	uint64_t carry, r0, diff;

	/* Zero the carry. */
	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		diff = (*a) - (*b);
		r0 = diff - carry;
		carry = ((*a) < (*b)) || (carry && !diff);
		(*c) = r0;
	}
	return carry;
}

__host__ __device__ uint32_t bn_subn_low_32(uint32_t * c, const uint32_t * a,
		const uint32_t * b, int size) {
	int i;
	uint32_t carry, r0, diff;

	/* Zero the carry. */
	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		diff = (*a) - (*b);
		r0 = diff - carry;
		carry = ((*a) < (*b)) || (carry && !diff);
		(*c) = r0;
	}
	return carry;
}

/**
 * [bn_sub1_low description]
 * @param  c     [description]
 * @param  a     [description]
 * @param  digit [description]
 * @param  size  [description]
 * @return       [description]
 */
__host__ __device__ uint64_t bn_sub1_low(uint64_t *c, const uint64_t *a, uint64_t digit, int size) {
	int i;
	uint64_t carry, r0;

	carry = digit;
	for (i = 0; i < size && carry; i++, c++, a++) {
		r0 = (*a) - carry;
		carry = (r0 > (*a));
		(*c) = r0;
	}
	for (; i < size; i++, a++, c++) {
		(*c) = (*a);
	}
	return carry;
}

__host__ __device__ uint32_t bn_sub1_low_32(uint32_t *c, const uint32_t *a, uint32_t digit, int size) {
	int i;
	uint32_t carry, r0;

	carry = digit;
	for (i = 0; i < size && carry; i++, c++, a++) {
		r0 = (*a) - carry;
		carry = (r0 > (*a));
		(*c) = r0;
	}
	for (; i < size; i++, a++, c++) {
		(*c) = (*a);
	}
	return carry;
}

/**
 * Accumulates a double precision digit in a triple register variable.
 *
 * @param[in,out] R2		- most significant word of the triple register.
 * @param[in,out] R1		- middle word of the triple register.
 * @param[in,out] R0		- lowest significant word of the triple register.
 * @param[in] A				- the first digit to multiply.
 * @param[in] B				- the second digit to multiply.
 */
// #ifdef __CUDA_ARCH__	

#define COMBA_STEP_BN_MUL_LOW(R2, R1, R0, A, B)														\
	uint64_t rHi = __umul64hi((uint64_t)(A) , (uint64_t)(B));										\
	uint64_t rLo = (uint64_t)(A) * (uint64_t)(B);													\
	uint64_t _r = (R1);																				\
	(R0) += rLo;																					\
	(R1) += (R0) < rLo;																				\
	(R2) += (R1) < _r;																				\
	(R1) += rHi;																					\
	(R2) += (R1) < rHi;

// #else

// #define COMBA_STEP_BN_MUL_LOW(R2, R1, R0, A, B)														\
// 	__uint128_t r = (__uint128_t)((uint64_t)(A))*(__uint128_t)((uint64_t)(B));						\
// 	uint64_t rHi = (r>>64);																			\
// 	uint64_t rLo = (r&0xffffffffffffffffL);															\
// 	uint64_t _r = (R1);																				\
// 	(R0) += rLo;																					\
// 	(R1) += (R0) < rLo;																				\
// 	(R2) += (R1) < _r;																				\
// 	(R1) += rHi;																					\
// 	(R2) += (R1) < rHi;
// #endif							

/**
 * Accumulates a single precision digit in a triple register variable.
 *
 * @param[in,out] R2		- most significant word of the triple register.
 * @param[in,out] R1		- middle word of the triple register.
 * @param[in,out] R0		- lowest significant word of the triple register.
 * @param[in] A				- the first digit to accumulate.
 */
#define COMBA_ADD(R2, R1, R0, A)											\
	cuyasheint_t __r = (R1);												\
	(R0) += (A);															\
	(R1) += (R0) < (A);														\
	(R2) += (R1) < __r;														\

/**
 * Multiplies two digit vectors of different sizes, with sizea > sizeb. 
 * Computes c = a * b. 
 * 
 * This function outputs as result only the digits between low and high,
 * 	inclusive, with high > sizea and low < sizeb.
 * @param c  [description]
 * @param a  [description]
 * @param sa [description]
 * @param b  [description]
 * @param sb [description]
 * @param l  [description]
 * @param h  [description]
 */
__device__ void bn_muld_low(cuyasheint_t * c, 
							const cuyasheint_t * a, 
							int sa,
							const cuyasheint_t * b, 
							int sb, 
							int l, 
							int h) {
	int i, j, ta;
	const cuyasheint_t *tmpa, *tmpb;
	cuyasheint_t r0, r1, r2;

	c += l;

	r0 = r1 = r2 = 0;
	for (i = l; i < sb; i++, c++) {
		tmpa = a;
		tmpb = b + i;
		for (j = 0; j <= i; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
	ta = 0;
	for (i = sb; i < sa; i++, c++) {
		tmpa = a + ++ta;
		tmpb = b + (sb - 1);
		for (j = 0; j < sb; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
	for (i = sa; i < h; i++, c++) {
		tmpa = a + ++ta;
		tmpb = b + (sb - 1);
		for (j = 0; j < sa - ta; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
}

/**
 * Multiplies two digit vectors of the same size. Computes c = a * b.
 * @param c    [description]
 * @param a    [description]
 * @param b    [description]
 * @param size [description]
 */
__device__ void bn_muln_low(cuyasheint_t *c,
							const cuyasheint_t *a,
							const cuyasheint_t *b,
							int size 
						){
	int i, j;
	const cuyasheint_t *tmpa, *tmpb;
	cuyasheint_t r0, r1, r2;

	r0 = r1 = r2 = 0;
	for (i = 0; i < size; i++, c++) {
		tmpa = a;
		tmpb = b + i;
		for (j = 0; j <= i; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
	for (i = 0; i < size; i++, c++) {
		tmpa = a + i + 1;
		tmpb = b + (size - 1);
		for (j = 0; j < size - (i + 1); j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}

}

/**
 * [bn_mod_barrt description]
 * @param c  [description]
 * @param a  [description]
 * @param sa [description]
 * @param m  [description]
 * @param sm [description]
 * @param u  [description]
 * @param su [description]
 */

__device__ void bn_mod_barrt(	cuyasheint_t *c, int *sc, const cuyasheint_t *a,const int sa,
								const cuyasheint_t * m,  int sm, const cuyasheint_t * u, int su
							) {

	if(bn_cmpn_low(a, m, sm) == CMP_LT)
		return;
	
	int mu;
	cuyasheint_t q[DSTD_BNT_WORDS_ALLOC],t[DSTD_BNT_WORDS_ALLOC],carry;

	#pragma unroll 200
	for(int i = 0; i < DSTD_BNT_WORDS_ALLOC; i++){
		q[i] = 0;
		t[i] = 0;
	}
	int sq, st;
	int i;

	mu = sm;
	sq = sa - (mu - 1);
	
	// bn_rsh
	for (i = 0; i < sq; i++) 
		q[i] = a[i + (mu - 1)];
	//
	
	if (sq > su) {
		// The first mu+1 coeficients are completely useless. 
		// There is a right shift after this.
		bn_muld_low(t, q, sq, u, su, mu, sq + su);
	} else {
		bn_muld_low(t, u, su, q, sq, mu - (su-sq), sq + su);
	}
	st = sq + su;

	// bn_trim
	while (st > 0 && t[st - 1] == 0)
		--(st);
	//

	// bn_rsh
	sq = st - (mu + 1);
	for (i = 0; i < sq; i++)
		q[i] = t[i + (mu + 1)];
	//
	
	if (sq > sm) 
		bn_muld_low(t, q, sq, m, sm, 0, sq + 1);
	else 
		bn_muld_low(t, m, sm, q, sq, 0, mu + 1);
	
	st = mu + 1;
	// bn_trim
	while (st > 0 && t[st - 1] == 0)
		st--;
	//
	
	// bn_mod_2b
	sq = mu + 1;
	for (i = 0; i < sq; i++) 
		q[i] = t[i];
	//
	
	// bn_mod_2b
	st = mu + 1;
	for (i = 0; i < sq; i++)
		t[i] = a[i];
	//

	carry = bn_subn_low(t, t, q, sq);
	// bn_trim
	while (st > 0 && t[st - 1] == 0)
		st--;
	//
	
	// If BN_NEG
	if (carry) {
		// bn_set_dig + bn_lsh
		sq = (mu + 1);
		for (i = 0; i < sq - 1; i++) {
			q[i] = 0;
		}
		q[sq - 1] = 1;
		//
		bn_subn_low(t, q, t, sq);
	}

	while (bn_cmpn_low(t, m, sm) != CMP_LT)
		bn_subn_low(t, t, m, sm);

	for (i = 0; i < st; i++)
		c[i] = t[i];
	*sc = st;
}

/**
 * [cuModN description]
 * @param c      [description]
 * @param a      [description]
 * @param NCoefs [description]
 * @param m      [description]
 * @param sm     [description]
 * @param u      [description]
 * @param su     [description]
 */
__global__ void cuModN(bn_t * c, bn_t * a, const int NCoefs,
		const cuyasheint_t * m, int sm, const cuyasheint_t * u, int su){
	/**
	 * This function should be executed with NCoefs threads
	 */
	const unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if(tid < NCoefs){
		bn_rem(c,a,NCoefs,m,get_used_index(m,sm)+1,u,get_used_index(u,su)+1);
		bn_zero_non_used(&a[tid]);
	}
}

/**
 * [callCuModN description]
 * @param c      [description]
 * @param a      [description]
 * @param NCoefs [description]
 * @param m      [description]
 * @param sm     [description]
 * @param u      [description]
 * @param su     [description]
 * @param stream [description]
 */
__host__ void callCuModN(bn_t * c, bn_t * a,int NCoefs,
		const cuyasheint_t * m, int sm, const cuyasheint_t * u, int su,
		hipStream_t stream){

	const int size = NCoefs;
	int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? 
			size/ADDBLOCKXDIM : 
			size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);

	cuModN<<<gridDim,blockDim,0,stream>>>(c,a,NCoefs,m,sm,u,su);
}
/////////
// CRT //
/////////

/**
 * @d_polyCRT - output: array of residual polynomials
 * @x - input: array of coefficients
 * @ N - input: qty of coefficients
 * @NPolis - input: qty of primes/residual polynomials
 */
__global__ void cuCRT(	cuyasheint_t *d_polyCRT,
						bn_t *x,
						const int used_coefs,
						const unsigned int N,
						const unsigned int NPolis
						){
	/**
	 * This function should be executed with used_coefs*NPolis threads. 
	 * Each thread computes one residue of one coefficient
	 *
	 * x should be an array of N elements
	 * d_polyCRT should be an array of N*NPolis elements
	 */
	
	/**
	 * tid: thread id
	 * cid: coefficient id
	 * rid: residue id
	 */
	const unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const unsigned int cid = tid % (N);
	const unsigned int rid = tid / (N);

	// x can be copied to shared memory!
	// 
	if(tid < N*NPolis)
		// Computes x mod pi
		d_polyCRT[cid + rid*N] = bn_mod1_low(	x[cid].dp,
												x[cid].used,
												CRTPrimesConstant[rid]
												);

}	

__device__ int get_used_index(const cuyasheint_t *u,int alloc){
	int i = 0;
	// int max = 0;


	for(i = alloc-1; i >= 0; i--)
	/*
	 * Profiling shows that the branch is cheaper
	 */ 
		if(u[i] != 0)
			break;
		// max = i*isNotZero(u[i])*isZero(max) + max*isNotZero(max);


	return i;
}

__global__ void cuPreICRT(	cuyasheint_t *inner_results,
							cuyasheint_t *inner_results_used,
							const cuyasheint_t *d_polyCRT,
							const unsigned int N,
							const unsigned int NPolis
						){

	/**
	 * This kernel has a very bad access pattern in inner_results
	 */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(tid < N*NPolis){
	 	const int cid = tid % N;
	 	const int rid = tid / N;


		// Get a prime
		cuyasheint_t pi = CRTPrimesConstant[rid];

		/**
		 * Inner
		 */
		cuyasheint_t x;		
		bn_64bits_mulmod(	&x,
							invMpis[rid],
							d_polyCRT[cid + rid*N],
							pi);

		// Adjust available words in inner_result
		// assert(inner_result.alloc >= Mpis[rid].used+1);
		// bn_grow_d(&inner_result,1);
		int carry;
		carry = bn_mul1_low(	&inner_results[tid*STD_BNT_WORDS_ALLOC],
						     	&Mpis[rid*STD_BNT_WORDS_ALLOC],
						     	x,
						     	Mpis_used[rid]);

		inner_results_used[tid] = Mpis_used[rid];

		/** 
		 * The frequency of branching at this point is so low that 
		 * doing this is faster than using some non-branch technic
		 */
		if(carry){
			inner_results[tid*STD_BNT_WORDS_ALLOC + Mpis_used[rid]] = carry;	
			inner_results_used[tid] += 1;			
		}
	}
}

// Div
/**
 * Divides a digit vector by another digit vector. 
 * Computes c = floor(a / b) and d = a mod b. 
 * 
 * The dividend and the divisor are destroyed inside the function.
 * 
 * @param c  [description]
 * @param d  [description]
 * @param a  [description]
 * @param sa [description]
 * @param b  [description]
 * @param sb [description]
 */
// __device__ void bn_divn_low( 	uint32_t *c, 
// 								uint32_t *d, 
// 								uint32_t *a, 
// 								int sa, 
// 								uint32_t *b, 
// 								int sb
// 							) {

__device__ void bn_rem(	bn_t *R, const bn_t *A,const int NCoefs,
								const cuyasheint_t * m,  int sm, const cuyasheint_t * u, int su
							) {

	/**
	 * Each thread handles one coefficient
	 */
	
	const int cid = threadIdx.x + blockDim.x*blockIdx.x;

	if(cid < NCoefs){
		cuyasheint_t *a = A[cid].dp;
		int sa = A[cid].used;

		// Remainder
		cuyasheint_t *rem = R[cid].dp;
		int sr = R[cid].used;

		// Quotient
		// cuyasheint_t quot[STD_BNT_WORDS_ALLOC];

		// bn_divn_low(quot, rem, a, sa, M, M_used);
		// if (bn_cmpn_low<cuyasheint_t>(a, quot, sa) == CMP_LT) {

			bn_mod_barrt( rem, &sr, a, sa, M, M_used, u, u_used );
			R[cid].used = sr;
		// }
	}

}

/**
 * cuICRT computes ICRT on GPU
 * @param poly      output: An array of coefficients 
 * @param d_polyCRT input: The CRT residues
 * @param N         input: Number of coefficients
 * @param NPolis    input: Number of residues
 */
__global__ void cuICRT(	bn_t *poly,
						const unsigned int N,
						const unsigned int NPolis,
						cuyasheint_t *inner_results,
						cuyasheint_t *inner_results_used
						){
	/**
	 * This function should be executed with N threads.
	 * Each thread j computes a Mpi*( invMpi*(value) % pi) and adds to poly[j]
	 */
	
	/**
	 * tid: thread id
	 * cid: coefficient id
	 * rid: residue id
	 */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const int cid = tid;
	
	 if(tid < N){
	 	bn_t coef = poly[cid];
	 	bn_zero(&coef); 

	 	for(unsigned int rid = 0; rid < NPolis;rid++){
		 	cuyasheint_t *inner_result = &inner_results[(cid + rid*N)*STD_BNT_WORDS_ALLOC];

			/**
			 * Accumulate
			 */
			int nwords = max_d(coef.used,inner_results_used[(cid + rid*N)]);
			cuyasheint_t carry = bn_addn_low(coef.dp, coef.dp, inner_result,nwords);
			coef.used = nwords;

			/* Equivalent to "If has a carry, add as last word" */
			coef.dp[coef.used] = carry;
			coef.used += (carry > 0);
 		}

		////////////////////////////////////////////////
		// Modular reduction of poly[cid] by M //
		////////////////////////////////////////////////
 	 	/**
 	 	 * At this point a thread i finished the computation of coefficient i
 	 	 */
 	 	poly[cid] = coef;
		bn_rem(	poly,
						poly,
						N,
						M,
						M_used,
						u,
						u_used);
    	bn_zero_non_used(&poly[cid]);
	 }

}
	


void callCRT(bn_t *coefs,const int used_coefs,cuyasheint_t *d_polyCRT,const int N, const int NPolis,hipStream_t stream){
	const int size = N*NPolis;

	if(size <= 0)
		return;

	hipError_t result;

	// Set all positions to 0
	result = hipMemsetAsync(d_polyCRT,0,size*sizeof(cuyasheint_t),stream);
    assert(result == hipSuccess);
	
	int blockSize;   // The launch configurator returned block size 
	// int minGridSize; // The minimum grid size needed to achieve the 
           			 // maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 

	// hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, cuCRT, 0, 0); 
	// blockSize = 32; // 0.11 ms
	// blockSize = 64; // 0.07 ms
	// blockSize = 128; // 0.06 ms
	// blockSize = 192; // 0.06 ms
	// blockSize = 256; // 0.06 ms
	blockSize = 512; // 0.07 ms

	gridSize = (size%blockSize == 0? 
			size/blockSize : 
			size/blockSize + 1);
	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);
	
	cuCRT<<<gridDim,blockDim,0,stream>>>(d_polyCRT,coefs,used_coefs,N,NPolis);
	result = hipGetLastError();
	assert(result == hipSuccess);
	// hipDeviceSynchronize();
	// assert(result == hipSuccess);

}

void callICRT(bn_t *coefs,cuyasheint_t *d_polyCRT,const int N, const int NPolis,hipStream_t stream){

	if(N <= 0)
		return;
	int blockSize;   // The launch configurator returned block size 
	// int minGridSize; // The minimum grid size needed to achieve the 
           			 // maximum occupancy for a full device launch 
	int gridSize;    // The actual grid size needed, based on input size 

	// hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, cuPreICRT, 0, 0); 
	// blockSize = 32; // 0.71 ms
	// blockSize = 64; // 0.58 ms
	// blockSize = 128; // 0.54 ms
	// blockSize = 192; // 0.54 ms
	// blockSize = 256; // 0.54 ms
	blockSize = 512; // 0.54 ms

	gridSize = ( N*NPolis % blockSize == 0? 
						N*NPolis/blockSize : 
						N*NPolis/blockSize + 1);
	cuPreICRT<<<gridSize,blockSize,0,stream>>> (CUDAFunctions::d_inner_results,
												CUDAFunctions::d_inner_results_used,
												d_polyCRT,
												N,
												NPolis
												);
	blockSize = 64;

	gridSize = ( N % blockSize == 0? 
						N/blockSize : 
						N/blockSize + 1);
	cuICRT<<<gridSize,blockSize,0,stream>>>(coefs,
											N,
											NPolis,
											CUDAFunctions::d_inner_results,
											CUDAFunctions::d_inner_results_used);
	hipError_t result = hipGetLastError();
	assert(result == hipSuccess);
	// hipDeviceSynchronize();
	// assert(result == hipSuccess);

}

__host__ void  CUDAFunctions::write_crt_primes(){

  #ifdef VERBOSE
  std::cout << "primes: "<< std::endl;
  for(unsigned int i = 0; i < Polynomial::CRTPrimes.size();i++)
    std::cout << Polynomial::CRTPrimes[i] << " ";
  std::cout << std::endl;
  #endif
  
  // Choose what memory will be used to story CRT Primes
  if(Polynomial::CRTPrimes.size() < MAX_PRIMES_ON_C_MEMORY){
    
    #ifdef VERBOSE
    std::cout << "Writting CRT Primes to GPU's constant memory" << std::endl;
    #endif

    hipStream_t stream;
    hipStreamCreate(&stream);

    /////////////////
    // Copy primes //
    /////////////////
    hipError_t result = hipMemcpyToSymbolAsync(HIP_SYMBOL( CRTPrimesConstant),
                                              &(Polynomial::CRTPrimes[0]),
                                              Polynomial::CRTPrimes.size()*sizeof(cuyasheint_t),
                                              0,
	                                           hipMemcpyHostToDevice,
                                              stream
                                            );
    assert(result == hipSuccess);

    ////////////
    // Copy M //
    ////////////
    
    bn_t h_M;
    
    get_words_host(&h_M,Polynomial::CRTProduct);
	assert(h_M.alloc >= STD_BNT_WORDS_ALLOC);
	result = hipMemcpyToSymbolAsync(HIP_SYMBOL(M),h_M.dp, h_M.used*sizeof(cuyasheint_t),0,hipMemcpyHostToDevice,stream);
    assert(result == hipSuccess);
	result = hipMemcpyToSymbolAsync(HIP_SYMBOL(M_used),&h_M.used, sizeof(int),0,hipMemcpyHostToDevice,stream);
    assert(result == hipSuccess);
    
    ////////////
    // Copy u //
    ////////////

    cuyasheint_t *h_u;
    bn_t d_u = get_reciprocal(Polynomial::CRTProduct);
    h_u = (cuyasheint_t*)malloc(d_u.alloc*sizeof(cuyasheint_t));
	// assert(d_u.alloc >= STD_BNT_WORDS_ALLOC);
    hipMemcpyAsync(h_u,d_u.dp,d_u.alloc*sizeof(cuyasheint_t),hipMemcpyDeviceToHost,stream);
    result = hipMemcpyToSymbolAsync(HIP_SYMBOL(u),h_u,d_u.alloc*sizeof(cuyasheint_t),0,hipMemcpyHostToDevice,stream);
    assert(result == hipSuccess);
	result = hipMemcpyToSymbolAsync(HIP_SYMBOL(u_used),&d_u.used, sizeof(int),0,hipMemcpyHostToDevice,stream);
    assert(result == hipSuccess);
    
    //////////////
    // Copy Mpi //
    //////////////    
    bn_t *h_Mpis;
    h_Mpis = (bn_t*) malloc( Polynomial::CRTPrimes.size()*sizeof(bn_t) );
    
    for(unsigned int i = 0; i < Polynomial::CRTPrimes.size();i++){
    	h_Mpis[i].alloc = 0;
    	get_words_host(&h_Mpis[i],Polynomial::CRTMpi[i]);
		result = hipMemcpyToSymbolAsync(HIP_SYMBOL(Mpis), h_Mpis[i].dp, STD_BNT_WORDS_ALLOC*sizeof(cuyasheint_t),i*STD_BNT_WORDS_ALLOC*sizeof(cuyasheint_t),hipMemcpyHostToDevice,stream);
		assert(result == hipSuccess);
		result = hipMemcpyToSymbolAsync(HIP_SYMBOL(Mpis_used),&h_Mpis[i].used, sizeof(int),i*sizeof(int),hipMemcpyHostToDevice,stream);
		assert(result == hipSuccess);
    }

    /////////////////
    // Copy InvMpi //
    /////////////////

	result = hipMemcpyToSymbolAsync(HIP_SYMBOL(invMpis),
								&Polynomial::CRTInvMpi[0],
								Polynomial::CRTPrimes.size()*sizeof(cuyasheint_t),
								0,
								hipMemcpyHostToDevice,
								stream
							);
    assert(result == hipSuccess);

    ////////////////////
    // Release memory //
    ////////////////////
	result = hipDeviceSynchronize();
    assert(result == hipSuccess);
    for(unsigned int i = 0; i < Polynomial::CRTPrimes.size();i++)
    	free(h_Mpis[i].dp);

    free(h_Mpis);
    free(h_M.dp);
    hipFree(d_u.dp);
  }else{
    throw "Too many primes.";
  }
}