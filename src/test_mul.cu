/**
 * cuYASHE
 * Copyright (C) 2015-2016 cuYASHE Authors
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <NTL/ZZ_pEX.h>

#include "polynomial.h"
#include "settings.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

int main(){
   int degree;
  Polynomial phi;
  ZZ_pX NTL_Phi;

  ////////////////
  // First test //
  ////////////////
  std::cout << "Prime size: " << CRTPRIMESIZE << std::endl;
  std::cout << "First test: " << std::endl;
  degree = 8;

  Polynomial::global_mod = conv<ZZ>("1171313591017775093490277364417"); // Defines default GF(q)
  Polynomial::BuildNthCyclotomic(&phi,degree);
  phi.set_mod(Polynomial::global_mod);
  Polynomial::global_phi = &phi;
// 
  srand (36251);

  Polynomial::gen_crt_primes(Polynomial::global_mod,degree);
  std::cout << "M = " << Polynomial::CRTProduct << std::endl;
  ZZ_p::init(Polynomial::global_mod);
  for(int i = 0; i <= phi.deg();i++){
    NTL::SetCoeff(NTL_Phi,i,conv<ZZ_p>(phi.get_coeff(i)));
  }
  ZZ_pE::init(NTL_Phi);

  Polynomial a,b,c;
  a.set_coeffs(8);
  a.set_coeff(0,1304768405);
  a.set_coeff(1,1912295543);
  a.set_coeff(2,584306160);
  a.set_coeff(3,476779113);
  a.set_coeff(4,2057832156);
  a.set_coeff(5,1274012458);
  a.set_coeff(6,1818971124);
  a.set_coeff(7,1943566066);

  b.set_coeffs(8);
  b.set_coeff(0,47396539);
  b.set_coeff(1,12477803);
  b.set_coeff(2,540722570);
  b.set_coeff(3,1743223311);
  b.set_coeff(4,1316384259);
  b.set_coeff(5,1278652539);
  b.set_coeff(6,635141178);
  b.set_coeff(7,577646167);

  c = a*b;
  
  std::cout << "a: " << a.to_string() << std::endl;
  std::cout << "b: " << b.to_string() << std::endl;
  std::cout << "a*b: " <<c.to_string() << std::endl;

  ZZ expected_result1[] = {to_ZZ("61841506593550295L"),
                             to_ZZ("106916833401539892L"),
                             to_ZZ("757073061970113119L"),
                             to_ZZ("3338412596642650852L"),
                             to_ZZ("5470565564447407791L"),
                             to_ZZ("5548103536799580232L"),
                             to_ZZ("6089003692894423880L"),
                             to_ZZ("7733984529407217518L"),
                             to_ZZ("8022978165266350588L"),
                             to_ZZ("9170489421306427724L"),
                             to_ZZ("8993977458886462143L"),
                             to_ZZ("6882188452282780506L"),
                             to_ZZ("4376377560524434132L"),
                             to_ZZ("2285160538342347456L"),
                             to_ZZ("1122693488336169022L")};
  c.normalize();
  assert(c.get_coeffs().size() == 15);

  for(int i = 0; i < c.get_coeffs().size();i++)
      assert(c.get_coeff(i) == expected_result1[i]);
  std::cout << "It works!\n" << std::endl;;

  /////////////////
  // Second test //
  /////////////////
  std::cout << std::endl << "Second test: " << std::endl;
  a.set_coeffs(8);
  a.set_coeff(0,1304768405);
  a.set_coeff(1,1912295543);
  a.set_coeff(2,584306160);
  a.set_coeff(3,476779113);
  a.set_coeff(4,2057832156);
  a.set_coeff(5,1274012458);
  a.set_coeff(6,1818971124);
  a.set_coeff(7,1943566066);
  
  b.set_coeffs(8);
  b.set_coeff(0,to_ZZ("13860721405712617825882419437"));
  b.set_coeff(1,to_ZZ("911979968073012785910226533"));
  b.set_coeff(2,to_ZZ("60134818568832314735148266017"));
  b.set_coeff(3,to_ZZ("30228208549061091796489949751"));
  b.set_coeff(4,to_ZZ("42496336825615196107672318132"));
  b.set_coeff(5,to_ZZ("2538357487121433195951809389"));
  b.set_coeff(6,to_ZZ("19401757472799351763805908821"));
  b.set_coeff(7,to_ZZ("68415359680988770693366249506"));

  c.set_coeffs();
  c = a*b;
  
  std::cout << "a: " << a.to_string() << std::endl;
  std::cout << "b: " << b.to_string() << std::endl;
  std::cout << "a*b: " <<c.to_string() << std::endl;

  ZZ expected_result2[] = { to_ZZ("18085031360681010249051172126355487985"),
                            to_ZZ("27695718415243509623513993478082759156"),
                            to_ZZ("88304891436672768443904444999435867224"),
                            to_ZZ("161577734953354918933760345835324042047"),
                            to_ZZ("177348142234080397165460901588496065374"),
                            to_ZZ("150446511525560453344956466169095450596"),
                            to_ZZ("219533428887126844535620298106253625669"),
                            to_ZZ("315527818352347626764175594181751346765"),
                            to_ZZ("380494624177053395698844189209855453911"),
                            to_ZZ("280450476250800780083801312965924383119"),
                            to_ZZ("211828603909665970332138017649830954350"),
                            to_ZZ("252717044998320508082871060193586798902"),
                            to_ZZ("127386722625350085506673181613116824226"),
                            to_ZZ("162154161142687163774421077602378932930"),
                            to_ZZ("132969771469154360046681933849550863396")};
  c.normalize();
  assert(c.get_coeffs().size() == 15);
  
  for(int i = 0; i < c.get_coeffs().size();i++)
      assert(c.get_coeff(i) == expected_result2[i]);
  std::cout << "It works!\n" << std::endl;;
  // a.release();
  // b.release();
  // c.release();

  hipDeviceReset();
}
